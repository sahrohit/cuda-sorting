#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <thrust/sort.h>

using namespace std;

#define CUDA_CHECK_ERROR
#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __cudaCheckError(__FILE__, __LINE__)
inline void __cudaSafeCall(hipError_t err,
  const char * file,
    const int line) {
  #ifdef CUDA_CHECK_ERROR
  #pragma warning(push)
  #pragma warning(disable: 4127)

  do {
    if (hipSuccess != err) {
      fprintf(stderr,
        "cudaSafeCall() failed at %s:%i : %s\n",
        file, line, hipGetErrorString(err));
      exit(-1);
    }
  } while (0);
  #pragma warning(pop)
  #endif
  return;
}

inline void __cudaCheckError(const char * file,
  const int line) {
  #ifdef CUDA_CHECK_ERROR
  #pragma warning(push)
  #pragma warning(disable: 4127)

  do {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
      fprintf(stderr,
        "cudaCheckError() failed at %s:%i : %s.\n",
        file, line, hipGetErrorString(err));
      exit(-1);
    }

    err = hipDeviceSynchronize();
    if (hipSuccess != err) {
      fprintf(stderr,
        "cudaCheckError() with sync failed at %s:%i : %s.\n",
        file, line, hipGetErrorString(err));
      exit(-1);
    }
  } while (0);
  #pragma warning(pop)
  #endif
  return;
}

int * makeRandArray(const int size,
  const int seed) {
  srand(seed);
  int * array = new int[size];
  for (int i = 0; i < size; i++) {
    array[i] = std::rand() % size;
  }
  return array;
}

__device__ inline void swap(int * x, int * y) {
  int temp = * x;
  * x = * y;
  * y = temp;
}

__device__ void bubbleSort(int arr[], int n) {
  int i, j;
  for (i = 0; i < n - 1; i++)
    for (j = 0; j < n - i - 1; j++)
      if (arr[j] > arr[j + 1])
        swap( & arr[j], & arr[j + 1]);
}

__global__ void matavgKernel(int a[], int n) {
  bubbleSort(a, n);
}

int main(int argc, char * argv[]) {
  int * array, * d_array;
  int size, seed;
  bool printSorted = false;

  if (argc < 4) {
    std::cerr << "usage: " <<
      argv[0] <<
      " [amount of random nums to generate] [seed value for rand]" <<
      " [1 to print sorted array, 0 otherwise]" <<
      std::endl;
    exit(-1);
  }

  {
    std::stringstream ss1(argv[1]);
    ss1 >> size;
  }

  {
    std::stringstream ss1(argv[2]);
    ss1 >> seed;
  }

  {
    int sortPrint;
    std::stringstream ss1(argv[3]);
    ss1 >> sortPrint;
    if (sortPrint == 1)
      printSorted = true;
  }

  array = (int * ) malloc(size * sizeof(int));

  array = makeRandArray(size, seed);

  hipEvent_t startTotal, stopTotal;
  float timeTotal;
  hipEventCreate( & startTotal);
  hipEventCreate( & stopTotal);
  hipEventRecord(startTotal, 0);

  CudaSafeCall(hipMalloc((void ** ) & d_array, size * sizeof(int)));

  hipMemcpy(d_array, array, size * sizeof(int), hipMemcpyHostToDevice);
  matavgKernel << < 1, 1 >>> (d_array, size - 1);

  hipMemcpy(array, d_array, size * sizeof(int), hipMemcpyDeviceToHost);

  hipEventRecord(stopTotal, 0);
  hipEventSynchronize(stopTotal);
  hipEventElapsedTime( & timeTotal, startTotal, stopTotal);
  hipEventDestroy(startTotal);
  hipEventDestroy(stopTotal);

  hipFree(d_array);

  std::cerr << "elapsed time: " << timeTotal / 1000.0 << std::endl;

  if (printSorted) {
    for (int i = 0; i < size; i++) {
      cout << array[i] << " ";
    }
  }
}